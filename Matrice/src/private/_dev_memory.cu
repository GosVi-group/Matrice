/**************************************************************************
This file is part of Matrice, an effcient and elegant C++ library.
Copyright(C) 2018, Zhilong(Dgelom) Su, all rights reserved.

This program is free software : you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful, but WITHOUT
ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.See the GNU General Public License for
more details.

You should have received a copy of the GNU General Public License
along with this program.If not, see <http://www.gnu.org/licenses/>.
**************************************************************************/
#include <complex>
#include <stdexcept>
#include "../../include/Matrice/util/_macros.h"
#include "../../include/Matrice/private/_devops.h"
#include "../../include/Matrice/private/_unified_memory.h"

#if (defined __enable_cuda__ && !defined __disable_cuda__)
#include <hip/hip_runtime.h>
#pragma warning(disable: 4715 4661 4224 4267 4244 4819 4199)

using std::size_t;
using std::complex;
using uchar = unsigned char;
MATRICE_PRIVATE_BEGIN
//<note> w is the columns, h is the rows </note>
template<typename _Scalar, typename = typename std::enable_if<std::is_scalar<_Scalar>::value>::type>
_Scalar* device_malloc(size_t& w, size_t h)
{
	hipError_t sts; _Scalar* dptr;
	switch (h)
	{
	case 1:
		sts = hipMalloc(&dptr, w * sizeof(_Scalar));
		break;
	default:
		size_t pitch = 0;
		sts = hipMallocPitch(&dptr, &pitch, w * sizeof(_Scalar), h);
		w = pitch;
		break;
	}
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
	return dptr;
}
template<typename _Scalar, typename = typename std::enable_if<std::is_scalar<_Scalar>::value>::type>
_Scalar* global_malloc(size_t N)
{
	_Scalar* dptr;
	auto sts = hipMallocManaged(&dptr, N * sizeof(_Scalar));
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
	return dptr;
}
//<note> w is the columns, h is the rows, p is the pitch size if pitched memory is used </note>
template<typename _Scalar, int _Opt, typename = typename std::enable_if<std::is_scalar<_Scalar>::value>::type>
void device_memcpy(_Scalar* hptr, _Scalar* dptr, size_t w, size_t h = 1, size_t p = 1)
{
	if (w == 1) std::swap(w, h);
	size_t hpitch = w * sizeof(_Scalar);
	hipError_t sts;
	if (_Opt == ::hipMemcpyHostToDevice) {
		switch (p)
		{
		case 1:
			sts = hipMemcpy(dptr, hptr, hpitch*h, hipMemcpyHostToDevice);
			break;
		default:
			sts = hipMemcpy2D(dptr, p, hptr, hpitch, hpitch, h, hipMemcpyHostToDevice);
			break;
		}
	}
	if (_Opt == ::hipMemcpyDeviceToHost) {
		switch (p)
		{
		case 1:
			sts = hipMemcpy(hptr, dptr, hpitch*h, ::hipMemcpyDeviceToHost);
			break;
		default:
			sts = hipMemcpy2D(hptr, hpitch, dptr, p, hpitch, h, ::hipMemcpyDeviceToHost);
			break;
		}
	}
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
}
template<typename _Scalar, typename = typename std::enable_if<std::is_scalar<_Scalar>::value>::type>
void device_free(_Scalar* dptr) { if (dptr) hipFree(dptr); }

#pragma region <!-- explicit intantiation -->
template int* device_malloc(size_t&, size_t);
template char* device_malloc(size_t&, size_t);
template bool* device_malloc(size_t&, size_t);
template float* device_malloc(size_t&, size_t);
template double* device_malloc(size_t&, size_t);
template unsigned char* device_malloc(size_t&, size_t);
template int* global_malloc(size_t);
template char* global_malloc(size_t);
template bool* global_malloc(size_t);
template float* global_malloc(size_t);
template double* global_malloc(size_t);
template unsigned char* global_malloc(size_t);
template void device_memcpy<int, 1>(int*, int*, size_t, size_t, size_t);
template void device_memcpy<int, 2>(int*, int*, size_t, size_t, size_t);
template void device_memcpy<char, 1>(char*, char*, size_t, size_t, size_t);
template void device_memcpy<char, 2>(char*, char*, size_t, size_t, size_t);
template void device_memcpy<bool, 1>(bool*, bool*, size_t, size_t, size_t);
template void device_memcpy<bool, 2>(bool*, bool*, size_t, size_t, size_t);
template void device_memcpy<float, 1>(float*, float*, size_t, size_t, size_t);
template void device_memcpy<float, 2>(float*, float*, size_t, size_t, size_t);
template void device_memcpy<double, 1>(double*, double*, size_t, size_t, size_t);
template void device_memcpy<double, 2>(double*, double*, size_t, size_t, size_t);
template void device_memcpy<unsigned char, 1>(unsigned char*, unsigned char*, size_t, size_t, size_t);
template void device_memcpy<unsigned char, 2>(unsigned char*, unsigned char*, size_t, size_t, size_t);
template void device_free(int*);
template void device_free(char*);
template void device_free(bool*);
template void device_free(float*);
template void device_free(double*);
template void device_free(unsigned char*);
#pragma endregion

MATRICE_PRIVATE_END

#pragma region <!-- Impl. for device_memcpy -->
template<typename T>
template<typename... _Args> MATRICE_GLOBAL
void dgelom::device::device_memcpy<T, 0>::impl(_Args ...args)
{
	return;
}
template<typename T>
template<typename... _Args> MATRICE_GLOBAL
void dgelom::device::device_memcpy<T, 1>::impl(_Args ...args)
{
	dgelom::privt::device_memcpy<T, option>(args...);
}
template void dgelom::device::device_memcpy<uchar, 1>
::impl(pointer, pointer, size_t, size_t, size_t);
template void dgelom::device::device_memcpy<float, 1>
::impl(pointer, pointer, size_t, size_t, size_t);
template void dgelom::device::device_memcpy<double, 1>
::impl(pointer, pointer, size_t, size_t, size_t);
template<typename T>
template<typename... _Args> MATRICE_GLOBAL
void dgelom::device::device_memcpy<T, 2>::impl(_Args ...args)
{
	dgelom::privt::device_memcpy<T, option>(args...);
}
template void dgelom::device::device_memcpy<uchar, 2>
::impl(pointer, pointer, size_t, size_t, size_t);
template void dgelom::device::device_memcpy<float, 2>
::impl(pointer, pointer, size_t, size_t, size_t);
template void dgelom::device::device_memcpy<double, 2>
::impl(pointer, pointer, size_t, size_t, size_t);
#pragma endregion

#pragma region <!-- unified_sync class implementation -->
MATRICE_PRIVATE_BEGIN
template<typename _Scalar, Loc _Host>
_Scalar* unified_sync<_Scalar, _Host, Loc::OnDevice, LINEAR>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _1)
{
	auto _Stat = hipMemcpy(_Dst, _Src, _Rows*_Cols * sizeof(_Scalar), ::hipMemcpyHostToDevice);
	if(_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
int* unified_sync<int, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
uchar* unified_sync<uchar, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnStack, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
uchar* unified_sync<uchar, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnHeap, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);

template<typename _Scalar, Loc _Host>
_Scalar* unified_sync<_Scalar, _Host, Loc::OnDevice, PITCHED>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _Pytes)
{
	auto _Stat = hipMemcpy2D(_Dst, _Pytes, _Src, _Cols * sizeof(_Scalar), _Rows * _Cols * sizeof(_Scalar), ::hipMemcpyHostToDevice);
	if (_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
uchar* unified_sync<uchar, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnStack, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
uchar* unified_sync<uchar, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnHeap, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);

template<typename _Scalar, Loc _Host>
_Scalar* unified_sync<_Scalar, Loc::OnDevice, _Host, LINEAR>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _1)
{
	auto _Stat = hipMemcpy(_Dst, _Src, _Rows *_Cols * sizeof(_Scalar), ::hipMemcpyDeviceToHost);
	if (_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnStack, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnHeap, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);

template<typename _Scalar, Loc _Host>
_Scalar* unified_sync<_Scalar, Loc::OnDevice, _Host, PITCHED>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _Pytes)
{
	auto _Stat = hipMemcpy2D(_Dst, _Pytes, _Src, _Cols * sizeof(_Scalar), _Rows * _Cols * sizeof(_Scalar), ::hipMemcpyDeviceToHost);
	if (_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnStack, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnHeap, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);

template<typename _Scalar>
_Scalar* unified_sync<_Scalar, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _1)
{
	auto _Stat = hipMemcpy(_Dst, _Src, _Rows *_Cols * sizeof(_Scalar), ::hipMemcpyDeviceToDevice);
	if (_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnDevice, LINEAR>::op(pointer, const_pointer, size_t, size_t, size_t);

template<typename _Scalar>
_Scalar* unified_sync<_Scalar, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer _Dst, const_pointer _Src, size_t _Rows, size_t _Cols, size_t _Pytes)
{
	auto _Stat = hipMemcpy2D(_Dst, _Pytes, _Src, _Cols * sizeof(_Scalar), _Rows * _Cols * sizeof(_Scalar), ::hipMemcpyDeviceToDevice);
	if (_Stat != hipSuccess)
#ifdef _DEBUG
		throw std::runtime_error(hipGetErrorString(_Stat));
#else
		return nullptr
#endif
	else return (_Dst);
}
uchar* unified_sync<uchar, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
float* unified_sync<float, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
double* unified_sync<double, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
int* unified_sync<int, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
char* unified_sync<char, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
bool* unified_sync<bool, Loc::OnDevice, Loc::OnDevice, PITCHED>::op(pointer, const_pointer, size_t, size_t, size_t);
MATRICE_PRIVATE_END
#pragma endregion

#endif
/**************************************************************************
This file is part of Matrice, an effcient and elegant C++ library.
Copyright(C) 2018, Zhilong(Dgelom) Su, all rights reserved.

This program is free software : you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful, but WITHOUT
ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
FITNESS FOR A PARTICULAR PURPOSE.See the GNU General Public License for
more details.

You should have received a copy of the GNU General Public License
along with this program.If not, see <http://www.gnu.org/licenses/>.
**************************************************************************/
#include <complex>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "../../include/Matrice/util/_macros.h"

#pragma warning(disable: 4715 4661 4224 4267 4244 4819 4199)

using std::size_t;
using std::complex;

MATRICE_PRIVATE_BEGIN

//<note> w is the columns, h is the rows </note>
template<typename _Scalar, typename = typename std::enable_if<std::is_literal_type<_Scalar>::value>::type>
_Scalar* device_malloc(_Scalar* dptr, size_t& w, size_t h)
{
	hipError_t sts;
	switch (h)
	{
	case 1:
		sts = hipMalloc(&dptr, w * sizeof(_Scalar));
		break;
	default:
		size_t pitch = 0;
		sts = hipMallocPitch(&dptr, &pitch, w * sizeof(_Scalar), h);
		w = pitch;
		break;
	}
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
	return dptr;
}
template<typename _Scalar, typename = typename std::enable_if<std::is_literal_type<_Scalar>::value>::type>
_Scalar* global_malloc(_Scalar* dptr, size_t N)
{
	auto sts = hipMallocManaged(&dptr, N * sizeof(_Scalar));
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
	return dptr;
}
//<note> w is the columns, h is the rows </note>
template<typename _Scalar, int _Opt, typename = typename std::enable_if<std::is_literal_type<_Scalar>::value>::type>
void device_memcpy(_Scalar* hptr, _Scalar* dptr, size_t w, size_t h = 1, size_t p = 1)
{
	if (w == 1) std::swap(w, h);
	size_t hpitch = w * sizeof(_Scalar);
	hipError_t sts;
	if (_Opt == ::hipMemcpyHostToDevice) {
		switch (p)
		{
		case 1:
			sts = hipMemcpy(dptr, hptr, hpitch*h, ::hipMemcpyHostToDevice);
			break;
		default:
			sts = hipMemcpy2D(dptr, p, hptr, hpitch, hpitch, h, ::hipMemcpyHostToDevice);
			break;
		}
	}
	if (_Opt == ::hipMemcpyDeviceToHost) {
		switch (p)
		{
		case 1:
			sts = hipMemcpy(hptr, dptr, hpitch*h, ::hipMemcpyDeviceToHost);
			break;
		default:
			sts = hipMemcpy2D(hptr, hpitch, dptr, p, hpitch, h, ::hipMemcpyDeviceToHost);
			break;
		}
	}
	if (sts != hipSuccess) throw std::runtime_error(hipGetErrorString(sts));
}
template<typename _Scalar, typename = typename std::enable_if<std::is_literal_type<_Scalar>::value>::type>
void device_free(_Scalar* dptr)
{
	if (dptr) hipFree(dptr);
}

#pragma region <!-- explicit intantiation -->
template int* device_malloc(int*, size_t&, size_t);
template char* device_malloc(char*, size_t&, size_t);
template bool* device_malloc(bool*, size_t&, size_t);
template float* device_malloc(float*, size_t&, size_t);
template double* device_malloc(double*, size_t&, size_t);
template unsigned char* device_malloc(unsigned char*, size_t&, size_t);
template complex<float>* device_malloc(complex<float>*, size_t&, size_t);
template complex<double>* device_malloc(complex<double>*, size_t&, size_t);
template int* global_malloc(int*, size_t);
template char* global_malloc(char*, size_t);
template bool* global_malloc(bool*, size_t);
template float* global_malloc(float*, size_t);
template double* global_malloc(double*, size_t);
template unsigned char* global_malloc(unsigned char*, size_t);
template complex<float>* global_malloc(complex<float>*, size_t);
template complex<double>* global_malloc(complex<double>*, size_t);
template void device_memcpy<int, 1>(int*, int*, size_t, size_t, size_t);
template void device_memcpy<int, 2>(int*, int*, size_t, size_t, size_t);
template void device_memcpy<char, 1>(char*, char*, size_t, size_t, size_t);
template void device_memcpy<char, 2>(char*, char*, size_t, size_t, size_t);
template void device_memcpy<bool, 1>(bool*, bool*, size_t, size_t, size_t);
template void device_memcpy<bool, 2>(bool*, bool*, size_t, size_t, size_t);
template void device_memcpy<float, 1>(float*, float*, size_t, size_t, size_t);
template void device_memcpy<float, 2>(float*, float*, size_t, size_t, size_t);
template void device_memcpy<double, 1>(double*, double*, size_t, size_t, size_t);
template void device_memcpy<double, 2>(double*, double*, size_t, size_t, size_t);
template void device_memcpy<unsigned char, 1>(unsigned char*, unsigned char*, size_t, size_t, size_t);
template void device_memcpy<unsigned char, 2>(unsigned char*, unsigned char*, size_t, size_t, size_t);
template void device_memcpy<complex<float>, 1>(complex<float>*, complex<float>*, size_t, size_t, size_t);
template void device_memcpy<complex<float>, 2>(complex<float>*, complex<float>*, size_t, size_t, size_t);
template void device_memcpy<complex<double>, 1>(complex<double>*, complex<double>*, size_t, size_t, size_t);
template void device_memcpy<complex<double>, 2>(complex<double>*, complex<double>*, size_t, size_t, size_t);
template void device_free(int*);
template void device_free(char*);
template void device_free(bool*);
template void device_free(float*);
template void device_free(double*);
template void device_free(unsigned char*);
template void device_free(complex<float>*);
template void device_free(complex<double>*);
#pragma endregion

MATRICE_PRIVATE_END